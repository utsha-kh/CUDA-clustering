#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <iomanip>
#include <cfloat>
#include <cmath>
#include <hip/hip_runtime.h>
#include <random>
#include "kmeans_gpu.h"
#include "parser.h"

int n;  // number of data points
int d;  // dimention of input data (usually 2, for 2D data)
int k;  // number of clusters

__device__ void d_getDistance(float* x1, float* x2, float *ret, int n, int d, int k);
__global__ void d_getMSE(float* dataPoints, int* labels, float* centeroids, float* ret, int n, int d, int k);
__global__ void d_assignDataPoints(float* dataPoints, int* labels, float* centeroids, int n, int d, int k);
__global__ void d_updateCenteroids(float* dataPoints, int* labels, float* centeroids, int* centeroids_sizes, int n, int d, int k);
__global__ void d_getWeights(float* dataPoints, float* centeroids, float* weights, int n, int d, int count);

// return L2 distance squared between 2 points
__device__ void d_getDistance(float* x1, float* x2, float *ret, int n, int d, int k){
	float dist = 0;
    for(int i = 0; i < d; i++){
        dist += (x2[i] - x1[i]) * (x2[i] - x1[i]);
    }
    *ret = dist; 
}

// return current Mean Squared Error value of all points. This is needed to detect convergence.
float getMSE(float** dataPoints, int* labels, float** centeroids){

    float error = 0;
    float* err = new float[n];  // distance between each dataPoints to centeroids
    float *d_dataPoints, *d_centeroids, *d_err; 
    int *d_labels;

    // Allocate memory on GPU
    hipMalloc(&d_dataPoints, sizeof(float) * n * d);
    hipMalloc(&d_labels, sizeof(int) * n);
    hipMalloc(&d_centeroids, sizeof(float) * k * d);
    hipMalloc(&d_err, sizeof(float) * n);    

    // Flattening both matrix to ease copying to GPU
    float* flattenDataPoints = new float[n * d];
    for(int i = 0; i < n; i++){
        for(int j = 0; j < d; j++){
            flattenDataPoints[i * d + j] = dataPoints[i][j];
        }
    }
    float* flattenCenteroids = new float[n * k];
    for(int i = 0; i < k; i++){
        for(int j = 0; j < d; j++){
            flattenCenteroids[i * d + j] = centeroids[i][j];
        }
    }

    // copy flattened data into GPU
    hipMemcpy(d_dataPoints, flattenDataPoints, sizeof(float) * n * d, hipMemcpyHostToDevice);
    hipMemcpy(d_labels, labels, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_centeroids, flattenCenteroids, sizeof(float) * k * d, hipMemcpyHostToDevice);
    
    // call the kernel function to compute RMSE values in parallel
    int block_size = n / THREAD_PER_BLOCK + (n % THREAD_PER_BLOCK != 0);
    d_getMSE<<<block_size, THREAD_PER_BLOCK>>>(d_dataPoints, d_labels, d_centeroids, d_err, n, d, k);
    hipDeviceSynchronize();

    // copy back the result from GPU to CPU
    hipMemcpy(err, d_err, sizeof(float) * n, hipMemcpyDeviceToHost);

    // Summing up computed errors. could be made faster by parallel reduction
    for(int i = 0; i < n; i++){
        error += err[i];
        //std::cout << "error[" << i << "] = " << err[i] << std::endl;
    }

    // deallocate GPU and CPU memory
    hipFree(d_dataPoints);
    hipFree(d_labels);
    hipFree(d_centeroids);
    hipFree(d_err);
    delete[] flattenDataPoints;
    delete[] flattenCenteroids;

    // return actual Mean of Squared Errors.
    return error / n;
}

// kernel of above function. NOTE: this is like helper of RMSE. The error values stored in err[] still needs to be summed up.
__global__ void d_getMSE(float* dataPoints, int* labels, float* centeroids, float* err, int n, int d, int k){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;
    d_getDistance(&dataPoints[id * d], &centeroids[labels[id] * d], &err[id], n, d, k); 
}

// Initialize the centeroids, based on k-means++ algorithm.
void initCenters(float** dataPoints, float** centeroids){
    
    float *d_dataPoints, *d_centeroids, *d_weights; 

    int count = 1;
    std::vector<float> weights_vec(n);
    float* weights = new float[n];

    std::cout << "Initializing centeroids basaed on k-means++ Algorighm..." << std::endl;

    std::random_device seedGenerator;
    std::mt19937 randomEngine(seedGenerator());
    std::uniform_int_distribution<> uniformRandom(0, n - 1);

    // 0. pick a random centeroid c1.
    int uniformLottery = uniformRandom(randomEngine);
    for(int i = 0; i < d; i++){
        centeroids[0][i] = dataPoints[uniformLottery][i];
    }
    // Allocate memory on GPU
    hipMalloc(&d_dataPoints, sizeof(float) * n * d);
    hipMalloc(&d_centeroids, sizeof(float) * k * d);
    hipMalloc(&d_weights, sizeof(float) * n);    

    // Flattening both matrix to ease copying to GPU
    float* flattenDataPoints = new float[n * d];
    for(int i = 0; i < n; i++){
        for(int j = 0; j < d; j++){
            flattenDataPoints[i * d + j] = dataPoints[i][j];
        }
    }
    float* flattenCenteroids = new float[n * k];
    for(int i = 0; i < k; i++){
        for(int j = 0; j < d; j++){
            flattenCenteroids[i * d + j] = centeroids[i][j];
        }
    }

    // copy flattened data into GPU
    hipMemcpy(d_dataPoints, flattenDataPoints, sizeof(float) * n * d, hipMemcpyHostToDevice);
    hipMemcpy(d_centeroids, flattenCenteroids, sizeof(float) * k * d, hipMemcpyHostToDevice);
    
    int block_size = n / THREAD_PER_BLOCK + (n % THREAD_PER_BLOCK != 0);

    while(count < k){
        // 1. for each data Points x, get Shortest Distance between x and a centeroid D(x)^2. This will be weight of that point. 
        d_getWeights<<<block_size, THREAD_PER_BLOCK>>>(d_dataPoints, d_centeroids, d_weights, n, d, count);
        hipMemcpy(weights, d_weights, sizeof(float) * n, hipMemcpyDeviceToHost); 
        // 2. pick a new cluster randomly from data points, with weighted sampling D(x)^2 / total D(x)^2
        for(int i = 0; i < n; i++){
            weights_vec[i] = weights[i];
        }
        std::discrete_distribution<int> weightedRandom(weights_vec.begin(), weights_vec.end());
        int weightedLottery = weightedRandom(randomEngine);
        for(int i = 0; i < d; i++){
            centeroids[count][i] = dataPoints[weightedLottery][i];
        }
        for(int i = 0; i < k; i++){
            for(int j = 0; j < d; j++){
                flattenCenteroids[i * d + j] = centeroids[i][j];
            }
        }   
        hipMemcpy(d_centeroids, flattenCenteroids, sizeof(float) * k * d, hipMemcpyHostToDevice);
        count++;
    }

    for(int i = 0; i < k; i++){
        for(int j = 0; j < d; j++){
            centeroids[i][j] = flattenCenteroids[i * d + j];
        }
    } 

    std::cout << "--Finished initialization!!" << std::endl;

    // deallocate GPU memory
    hipFree(d_dataPoints);
    hipFree(d_weights);
    hipFree(d_centeroids);
    delete[] flattenDataPoints;
    delete[] flattenCenteroids;
    delete[] weights; 

}

// kernel of above function
__global__ void d_getWeights(float* dataPoints, float* centeroids, float* weights, int n, int d, int count){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    float minDistance = FLT_MAX;
    float dist_i = 0;
    // find the closest centeroid from this dataPoint (&dataPoint[id * d]), and store the distance
    for(int i = 0; i < count; i++){
        d_getDistance(&dataPoints[id * d], &centeroids[i * d], &dist_i, n, d, count);
        if(dist_i < minDistance){
            minDistance = dist_i;
        }
    }
    weights[id] = minDistance;
}

// Assign each data point to the closest centeroid, and store the result in *labels
void assignDataPoints(float** dataPoints, int* labels, float** centeroids){

    float *d_dataPoints, *d_centeroids; 
    int *d_labels;

    // Allocate memory on GPU
    hipMalloc(&d_dataPoints, sizeof(float) * n * d);
    hipMalloc(&d_labels, sizeof(int) * n);
    hipMalloc(&d_centeroids, sizeof(float) * k * d);    

    // Flattening both matrix to ease copying to GPU
    float* flattenDataPoints = new float[n * d];
    for(int i = 0; i < n; i++){
        for(int j = 0; j < d; j++){
            flattenDataPoints[i * d + j] = dataPoints[i][j];
        }
    }
    float* flattenCenteroids = new float[n * k];
    for(int i = 0; i < k; i++){
        for(int j = 0; j < d; j++){
            flattenCenteroids[i * d + j] = centeroids[i][j];
        }
    }

    // copy flattened data into GPU
    hipMemcpy(d_dataPoints, flattenDataPoints, sizeof(float) * n * d, hipMemcpyHostToDevice);
    hipMemcpy(d_labels, labels, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_centeroids, flattenCenteroids, sizeof(float) * k * d, hipMemcpyHostToDevice);
    
    // call the kernel function to compute RMSE values in parallel
    int block_size = n / THREAD_PER_BLOCK + (n % THREAD_PER_BLOCK != 0);
    d_assignDataPoints<<<block_size, THREAD_PER_BLOCK>>>(d_dataPoints, d_labels, d_centeroids, n, d, k);
    hipDeviceSynchronize();

    // copy back the result from GPU to CPU
    hipMemcpy(labels, d_labels, sizeof(int) * n, hipMemcpyDeviceToHost);

    // deallocate GPU memory
    hipFree(d_dataPoints);
    hipFree(d_labels);
    hipFree(d_centeroids);
    delete[] flattenDataPoints;
    delete[] flattenCenteroids;

}

// kernal of above function
__global__ void d_assignDataPoints(float* dataPoints, int* labels, float* centeroids, int n, int d, int k){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;
    int closest = 0;
    float minDistance = FLT_MAX;
    float dist_i = 0;
    // find the closest centeroid (centeroids[closest]) from this dataPoint (&dataPoint[id * d])
    for(int i = 0; i < k; i++){
        d_getDistance(&dataPoints[id * d], &centeroids[i * d], &dist_i, n, d, k);
        if(dist_i < minDistance){
            closest = i;
            minDistance = dist_i;
        }
    }
    labels[id] = closest;
}

// add two vectors
__global__ void d_addVector(float* x1, float* x2, float* ret){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= 1000) return;
    ret[id] = x1[id] + x2[id];
}

void addVector(float* x1, float* x2, float* ret){
    for(int i = 0 ; i < d; i++){
        ret[i] = x1[i] + x2[i];
    }
}

// divide vector by scaler
__global__ void d_divideVector(float* x, int s, float* ret){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= 1000) return;
    ret[id] = x[id] / (float)s;
}

void divideVector(float* x, int s, float* ret){
    for(int i = 0; i < d; i++){
        ret[i] = x[i] / (float)s;
    }
}

// Update each center of sets u_i to the average of all data points who belong to that set
void updateCenteroids(float** dataPoints, int* labels, float** centeroids){

    float *d_dataPoints, *d_centeroids; 
    int *d_labels;
    int *d_centeroids_sizes;
    int *centeroids_sizes = new int[k]; // how many data points are avergaged to count each centeroid

    // Allocate memory on GPU
    hipMalloc(&d_dataPoints, sizeof(float) * n * d);
    hipMalloc(&d_labels, sizeof(int) * n);
    hipMalloc(&d_centeroids, sizeof(float) * k * d);    
    hipMalloc(&d_centeroids_sizes, sizeof(int) * k);

    // Flattening both matrix to ease copying to GPU
    float* flattenDataPoints = new float[n * d];
    for(int i = 0; i < n; i++){
        for(int j = 0; j < d; j++){
            flattenDataPoints[i * d + j] = dataPoints[i][j];
        }
    }
    float* flattenCenteroids = new float[n * k];
    for(int i = 0; i < k; i++){
        for(int j = 0; j < d; j++){
            flattenCenteroids[i * d + j] = 0; // reset values before passing to GPU. centeroids[i][j];
        }
    }
    for(int i = 0; i < k; i++){
        centeroids_sizes[i] = 0;    // reset values before passing to GPU. 
    }

    // copy flattened data into GPU
    hipMemcpy(d_dataPoints, flattenDataPoints, sizeof(float) * n * d, hipMemcpyHostToDevice);
    hipMemcpy(d_labels, labels, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_centeroids, flattenCenteroids, sizeof(float) * k * d, hipMemcpyHostToDevice);
    hipMemcpy(d_centeroids_sizes, centeroids_sizes, sizeof(int) * k, hipMemcpyHostToDevice);

    // call the kernel function to compute RMSE values in parallel
    // Here, I'm using 2D threads to parallelize double for loop so O(n * k) becomes O(1).
    int block_row_size = THREAD_PER_BLOCK / k;  //e.g. if k == 3, this is 1024 / 3 = 341
    int block_col_size = k;
    int number_of_blocks = n / block_row_size + (n % block_row_size != 0);
        // when k = 3, a block's shape will be 341 x 3 x 1, so it will have 1023 threads in a block, with 2 index per block
    dim3 block_shape(block_row_size, block_col_size, 1);
        // in total, there will be n x k threads. 
    d_updateCenteroids<<< number_of_blocks, block_shape >>>(d_dataPoints, d_labels, d_centeroids, d_centeroids_sizes, n, d, k);
    hipDeviceSynchronize();

    // copy back the result from GPU to CPU
    hipMemcpy(flattenCenteroids, d_centeroids, sizeof(float) * k * d, hipMemcpyDeviceToHost);
    hipMemcpy(centeroids_sizes, d_centeroids_sizes, sizeof(int) * k, hipMemcpyDeviceToHost);

    // put the flattened form of centeroids back to matrix form
    for(int i = 0; i < k; i++){
        for(int j = 0; j < d; j++){
            centeroids[i][j] = flattenCenteroids[i * d + j];
        }
    }

    // the centeroids computed by GPU was just sum of all points. Still needs to be divided by counts. 
    for(int i = 0; i < k; i++){
        divideVector(centeroids[i], centeroids_sizes[i], centeroids[i]);
    }

    // deallocate GPU memory
    hipFree(d_dataPoints);
    hipFree(d_labels);
    hipFree(d_centeroids);
    hipFree(d_centeroids_sizes);
    delete[] flattenDataPoints;
    delete[] flattenCenteroids;
    delete[] centeroids_sizes;

}

// kernel of above function
__global__ void d_updateCenteroids(float* dataPoints, int* labels, float* centeroids, int* centeroids_sizes, int n, int d, int k){
        // Here, each thread has a 2D index (id_x, id_y). Range: id_x = [0, n -1], id_y = [0, k -1]. 
    int id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int id_y = blockIdx.y * blockDim.y + threadIdx.y;
    if(id_x >= n || id_y >= k) return;
        // if, the data Point at id_x belongs to the id_y-th centeroid
    if(labels[id_x] == id_y){
        for(int i = 0; i < d; i++){
            atomicAdd(&centeroids[id_y * d + i], dataPoints[id_x * d + i]);
        }
        atomicAdd(&centeroids_sizes[id_y], 1);
    }
}

// need this to check convergence
float myAbs(float a, float b){
    if(a > b)
        return a - b;
    else
        return b - a;
}

// Checks convergence (d/dt < 0.5%)
// the CONVERGENCE_RATE is defined in kmeans_gpu.h
bool hasConverged(float prevError, float currentError){
    return (prevError - currentError) / prevError < CONVERGENCE_RATE || (prevError - currentError) / prevError > -CONVERGENCE_RATE;
}

// Calling this function will do everything for the user
void kMeansClustering(float** dataPoints, int* labels, int n_, int d_, int k_){
    n = n_; d = d_; k = k_; // copy arguments to global variables

    float** centeroids = new float*[k];
    for(int i = 0; i < k; i++){
        centeroids[i] = new float[d];
    }

    initCenters(dataPoints, centeroids);

    int iterations = 0;
    float previousError = FLT_MAX;
    float currentError = 0;
    while(iterations < MAX_ITERATIONS){    
        assignDataPoints(dataPoints, labels, centeroids);
        updateCenteroids(dataPoints, labels, centeroids);
        currentError = getMSE(dataPoints, labels, centeroids);
        std::cout << "(iteration" << iterations << ") Total Error Now: " << std::setprecision(6) << currentError << std::endl;
        if(hasConverged(previousError, currentError)) break;
        previousError = currentError;
        iterations++;
        }
    std::cout << "--Finished. # of iterations: " << iterations << std::endl;

    // free memory
    for(int i = 0; i < k; i++){
        delete[] centeroids[i];
    }
    delete[] centeroids;
}
