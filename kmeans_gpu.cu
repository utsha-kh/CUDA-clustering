#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <iomanip>
#include <cfloat>
#include <cmath>
#include <hip/hip_runtime.h>
#include "kmeans_gpu.h"
#include "parser.h"

int n;  // number of data points
int d;  // dimention of input data (usually 2, for 2D data)
int k;  // number of clusters

__device__ void d_getDistance(float* x1, float* x2, float *ret, int n, int d, int k);
__global__ void d_getMSE(float* dataPoints, int* labels, float* centeroids, float* ret, int n, int d, int k);
__global__ void d_assignDataPoints(float* dataPoints, int* labels, float* centeroids, int n, int d, int k);

// return L2 distance between 2 points
__device__ void d_getDistance(float* x1, float* x2, float *ret, int n, int d, int k){
	float dist = 0;
    for(int i = 0; i < d; i++){
        dist += (x2[i] - x1[i]) * (x2[i] - x1[i]);
    }
    *ret = dist; 
}

// return current Mean Squared Error value of all points. This is needed to detect convergence, but not essential in k-means algorithm.
float getMSE(float** dataPoints, int* labels, float** centeroids){

    float error = 0;
    float* err = new float[n];  // distance between each dataPoints to centeroids
    float *d_dataPoints, *d_centeroids, *d_err; 
    int *d_labels;

    // Allocate memory on GPU
    hipMalloc(&d_dataPoints, sizeof(float) * n * d);
    hipMalloc(&d_labels, sizeof(int) * n);
    hipMalloc(&d_centeroids, sizeof(float) * k * d);
    hipMalloc(&d_err, sizeof(float) * n);    

    // Flattening both matrix to ease copying to GPU
    float* flattenDataPoints = new float[n * d];
    for(int i = 0; i < n; i++){
        for(int j = 0; j < d; j++){
            flattenDataPoints[i * d + j] = dataPoints[i][j];
        }
    }
    float* flattenCenteroids = new float[n * k];
    for(int i = 0; i < k; i++){
        for(int j = 0; j < d; j++){
            flattenCenteroids[i * d + j] = centeroids[i][j];
        }
    }

    // copy flattened data into GPU
    hipMemcpy(d_dataPoints, flattenDataPoints, sizeof(float) * n * d, hipMemcpyHostToDevice);
    hipMemcpy(d_labels, labels, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_centeroids, flattenCenteroids, sizeof(float) * k * d, hipMemcpyHostToDevice);
    
    // call the kernel function to compute RMSE values in parallel
    int block_size = n / THREAD_SIZE + (n % THREAD_SIZE != 1);
    d_getMSE<<<block_size, THREAD_SIZE>>>(d_dataPoints, d_labels, d_centeroids, d_err, n, d, k);
    hipDeviceSynchronize();

    // copy back the result from GPU to CPU
    hipMemcpy(err, d_err, sizeof(float) * n, hipMemcpyDeviceToHost);

    // Summing up computed errors. could be made faster by parallel reduction
    for(int i = 0; i < n; i++){
        error += err[i];
        //std::cout << "error[" << i << "] = " << err[i] << std::endl;
    }

    // deallocate GPU and CPU memory
    hipFree(d_dataPoints);
    hipFree(d_labels);
    hipFree(d_centeroids);
    hipFree(d_err);
    delete[] flattenDataPoints;
    delete[] flattenCenteroids;

    // return actual Root Mean of Squared Errors.
    return error / n;
}

// kernel of above function. NOTE: this is like helper of RMSE. The error values stored in err[] still needs to be summed up.
__global__ void d_getMSE(float* dataPoints, int* labels, float* centeroids, float* err, int n, int d, int k){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;
    d_getDistance(&dataPoints[id * d], &centeroids[labels[id] * d], &err[id], n, d, k); 
}

// initialize each center values u_i to a randomly chosen data point
void initCenters(float** dataPoints, float** centeroids){
    // Each center u[i] should be a random data point x[j], but 
    // generating a non-repeated random number isn't straightforward
    // so I'll do it later
    for(int i = 0; i < k; i++){
        for(int j = 0; j < d; j++){
            centeroids[i][j] = dataPoints[i][j];
        }
    } 
}

// Assign each data point to the closest centeroid, and store the result in *labels
void assignDataPoints(float** dataPoints, int* labels, float** centeroids){

    float *d_dataPoints, *d_centeroids; 
    int *d_labels;

    // Allocate memory on GPU
    hipMalloc(&d_dataPoints, sizeof(float) * n * d);
    hipMalloc(&d_labels, sizeof(int) * n);
    hipMalloc(&d_centeroids, sizeof(float) * k * d);    

    // Flattening both matrix to ease copying to GPU
    float* flattenDataPoints = new float[n * d];
    for(int i = 0; i < n; i++){
        for(int j = 0; j < d; j++){
            flattenDataPoints[i * d + j] = dataPoints[i][j];
        }
    }
    float* flattenCenteroids = new float[n * k];
    for(int i = 0; i < k; i++){
        for(int j = 0; j < d; j++){
            flattenCenteroids[i * d + j] = centeroids[i][j];
        }
    }

    // copy flattened data into GPU
    hipMemcpy(d_dataPoints, flattenDataPoints, sizeof(float) * n * d, hipMemcpyHostToDevice);
    hipMemcpy(d_labels, labels, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_centeroids, flattenCenteroids, sizeof(float) * k * d, hipMemcpyHostToDevice);
    
    // call the kernel function to compute RMSE values in parallel
    int block_size = n / THREAD_SIZE + (n % THREAD_SIZE != 1);
    d_assignDataPoints<<<block_size, THREAD_SIZE>>>(d_dataPoints, d_labels, d_centeroids, n, d, k);
    hipDeviceSynchronize();

    // copy back the result from GPU to CPU
    hipMemcpy(labels, d_labels, sizeof(int) * n, hipMemcpyDeviceToHost);

    // deallocate GPU memory
    hipFree(d_dataPoints);
    hipFree(d_labels);
    hipFree(d_centeroids);
    delete[] flattenDataPoints;
    delete[] flattenCenteroids;
}

// kernal of above function
__global__ void d_assignDataPoints(float* dataPoints, int* labels, float* centeroids, int n, int d, int k){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;
    int closest = 0;
    float minDistance = FLT_MAX;
    float dist_i = 0;
    // find the closest centeroid (centeroids[closest]) from this dataPoint (&dataPoint[id * d])
    for(int i = 0; i < k; i++){
        d_getDistance(&dataPoints[id * d], &centeroids[i * d], &dist_i, n, d, k);
        if(dist_i < minDistance){
            closest = i;
            minDistance = dist_i;
        }
    }
    labels[id] = closest;
}

// add two vectors
__global__ void d_addVector(float* x1, float* x2, float* ret){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= 1000) return;
    ret[id] = x1[id] + x2[id];
}

void addVector(float* x1, float* x2, float* ret){
    for(int i = 0 ; i < d; i++){
        ret[i] = x1[i] + x2[i];
    }
}

// divide vector by scaler
__global__ void d_divideVector(float* x, int s, float* ret){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= 1000) return;
    ret[id] = x[id] / (float)s;
}

void divideVector(float* x, int s, float* ret){
    for(int i = 0; i < d; i++){
        ret[i] = x[i] / (float)s;
    }
}

// Update each center of sets u_i to the average of all data points who belong to that set
void updateCenteroids(float** dataPoints, int* labels, float** centeroids){
    int count = 0;
    for(int i = 0; i < k; i++){
        float* sum = new float[d];
        for(int l = 0; l < d; l++) sum[l] = 0;
        for(int j = 0; j < n; j++){
            if(labels[j] == i){
                addVector(sum, dataPoints[j], sum);
                count++;
            }
        }
        divideVector(sum, count, centeroids[i]);
        delete[] sum;
    }
}

// kernel of above function
__global__ void d_updateCenters(float** dataPoints, int* labels, float** centeroids){
    return;
}


float myAbs(float a, float b){
    if(a > b)
        return a - b;
    else
        return b - a;
}

// Checks convergence (d/dt < 0.5%)
bool hasConverged(float prevError, float currentError){
    return myAbs(prevError, currentError) / prevError < 0.005;
}

// Calling this function will do everything for the user
void kMeansClustering(float** dataPoints, int* labels, int n_, int d_, int k_){
    n = n_; d = d_; k = k_;

    float** centeroids = new float*[k];
    for(int i = 0; i < k; i++){
        centeroids[i] = new float[d];
    }

    initCenters(dataPoints, centeroids);

    int iterations = 0;
    float previousError = FLT_MAX;
    float currentError = 0;
    while(iterations < MAX_ITERATIONS){    
        assignDataPoints(dataPoints, labels, centeroids);
        updateCenteroids(dataPoints, labels, centeroids);
        currentError = getMSE(dataPoints, labels, centeroids);
        if(hasConverged(previousError, currentError)) break;
        previousError = currentError;
        iterations++;
        std::cout << "Total Error Now: " << std::setprecision(6) << currentError << std::endl;
    }
    std::cout << "# of iterations: " << iterations << std::endl;

    // free memory
    for(int i = 0; i < k; i++){
        delete[] centeroids[i];
    }
    delete[] centeroids;
}

